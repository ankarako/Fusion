#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dims, rtLaunchDim, );
///=======================================
///	Miss program for 360 environment maps
///=======================================
rtTextureSampler<float4, 2> EnvMapTexSampler;
RT_PROGRAM void EnvMapMiss()
{
	//unsigned int idx = launch_index.x + launch_index.y;
	//float theta = atan2f(ray.direction.x, ray.direction.z);
	//float phi	= M_PIf * 0.5f - acosf(ray.direction.y);
	//float u		= (theta + M_PIf) * (0.5f * M_1_PIf);
	//float v		= 0.5f * (1.0f + sinf(phi));
	float u = static_cast<float>(launch_index.x) / static_cast<float>(launch_dims.x);
	float v = static_cast<float>(launch_index.y) / static_cast<float>(launch_dims.y);
	float4 color = tex2D(EnvMapTexSampler, u, v);
	// if (idx == 0)
	// 	printf("color: (%f, %f, %f, %f)\n", color.x, color.y, color.x, color.w);
	prd_radiance.Result = fu::rt::make_color(optix::make_float3(color));
	// if (idx == 0)
	// 	printf("color: (%u, %u, %u, %u)\n", prd_radiance.Result.x, prd_radiance.Result.y, prd_radiance.Result.x, prd_radiance.Result.w);
}