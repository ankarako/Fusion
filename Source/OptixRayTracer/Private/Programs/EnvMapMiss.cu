#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
///=======================================
///	Miss program for 360 environment maps
///=======================================
rtTextureSampler<float4, 2> EnvMapTexSampler;
RT_PROGRAM void EnvMapMiss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi	= M_PIf * 0.5f - acosf(ray.direction.y);
	float u		= (theta + M_PIf) * (0.5f * M_1_PIf);
	float v		= 0.5f * (1.0f + sinf(phi));
	prd_radiance.Result = fu::rt::make_color(make_float3(tex2D(EnvMapTexSampler, u, v)));
}