#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
///=======================================
///	Miss program for 360 environment maps
///=======================================
rtTextureSampler<uchar4, 2> EnvMapTexSampler;
RT_PROGRAM void EnvMapMiss()
{
	unsigned int idx = launch_index.x + launch_index.y;
	if (idx == 0)
	{
		printf("current launch idx: %u\n", idx);
	}
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi	= M_PIf * 0.5f - acosf(ray.direction.y);
	float u		= (theta + M_PIf) * (0.5f * M_1_PIf);
	float v		= 0.5f * (1.0f + sinf(phi));
	
	prd_radiance.Result = tex2D(EnvMapTexSampler, u, v);
}