#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
///=======================================
///	Miss program for 360 environment maps
///=======================================
rtTextureSampler<float4, 2> EnvMapTexSampler;
RT_PROGRAM void EnvMapMiss()
{
	unsigned int idx = launch_index.x + launch_index.y;
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi	= M_PIf * 0.5f - acosf(ray.direction.y);
	float u		= (theta + M_PIf) * (0.5f * M_1_PIf);
	float v		= 0.5f * (1.0f + sinf(phi));
	float4 color = tex2D(EnvMapTexSampler, u, v);
	// if (idx == 0)
	// 	printf("color: (%f, %f, %f, %f)\n", color.x, color.y, color.x, color.w);
	prd_radiance.Result = fu::rt::make_color(optix::make_float3(color));
	// if (idx == 0)
	// 	printf("color: (%u, %u, %u, %u)\n", prd_radiance.Result.x, prd_radiance.Result.y, prd_radiance.Result.x, prd_radiance.Result.w);
}