#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float,  scene_epsilon, , );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void EnvMapRaygen()
{
	optix::size_t2 screen_size = output_buffer.size();
	optix::float2 d = optix::make_float2(launch_index) / optix::make_float2(screen_size) * optix::make_float2(2.0f * M_PIf, M_PIf) + optix::make_float2(M_PIf, 0);
	optix::float3 angle = optix::make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.y) * sin(d.y));
	optix::float3 ray_origin = eye;
	optix::float3 ray_direction = optix::normalize(angle.x * optix::normalize(U) + angle.y * optix::normalize(V) + angle.z * optix::normalize(W));
	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	fu::rt::PerRayData_Radiance prd;
	prd.Importance = 1.0f;
	prd.Depth = 0;

	output_buffer[launch_index] = prd.Result;
}