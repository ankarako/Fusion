#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtDeclareVariable(optix::float3, solid_color, , );
rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );

RT_PROGRAM void solid_color_closest_hit()
{
	prd_radiance.Result = fu::rt::make_color(solid_color);
}

RT_PROGRAM void solid_color_any_hit()
{
	prd_radiance.Result = fu::rt::make_color(solid_color);
}