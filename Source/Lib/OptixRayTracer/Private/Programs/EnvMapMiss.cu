#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dims, rtLaunchDim, );
///=======================================
///	Miss program for 360 environment maps
///=======================================
rtTextureSampler<float4, 2> EnvMapTexSampler;
RT_PROGRAM void EnvMapMiss()
{
	float u = static_cast<float>(launch_index.x) / static_cast<float>(launch_dims.x);
	float v = static_cast<float>(launch_index.y) / static_cast<float>(launch_dims.y);
	float4 color = tex2D(EnvMapTexSampler, u, v);
	prd_radiance.Result = fu::rt::make_color(optix::make_float3(color));
}