#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>

rtBuffer<float3> vertex_buffer;
rtBuffer<uint3> tindex_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(optix::float3, geometric_normal, attribute GeometricNormal, );
rtDeclareVariable(optix::float3, shading_normal, attribute ShadingNormal, );
rtDeclareVariable(optix::float3, back_hit_point, attribute BackHitPoint, );
rtDeclareVariable(optix::float3, front_hit_point, attribute FrontHitPoint, );
///=============
/// Intersection
///=============
RT_PROGRAM void triangle_mesh_intersect(int primIdx)
{
	const uint3 vertexIdx = tindex_buffer[primIdx];
	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];

	float3 normal;
	float t;
	float beta;
	float gamma;
	if (optix::intersect_triangle(ray, v0, v1, v2, normal, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			geometric_normal = optix::normalize(normal);
			shading_normal = geometric_normal;
			rtPotentialIntersection(0);
		}
	}
}
///==============
/// BBox Program
///==============
RT_PROGRAM void triangle_mesh_bounds(int primIdx, float result[6])
{
	const uint3 vertexIdx = tindex_buffer[primIdx];

	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];
	const float area = optix::length(optix::cross(v1 - v0, v2 - v0));
	
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
	{
		aabb->invalidate();
	}
}