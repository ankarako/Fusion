#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>

rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<uint3> tindex_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(optix::float3, geometric_normal, attribute GeometricNormal, );
rtDeclareVariable(optix::float3, shading_normal, attribute ShadingNormal, );
rtDeclareVariable(optix::float3, back_hit_point, attribute BackHitPoint, );
rtDeclareVariable(optix::float3, front_hit_point, attribute FrontHitPoint, );
///=============
/// Intersection
///=============
RT_PROGRAM void triangle_mesh_intersect(int primIdx)
{
	const uint3 vertexIdx = tindex_buffer[primIdx];
	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];

	float3 normal;
	float t;
	float beta;
	float gamma;
	if (optix::intersect_triangle(ray, v0, v1, v2, normal, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			geometric_normal = optix::normalize(normal);
			optix::float3 n0 = normal_buffer[vertexIdx.x];
			optix::float3 n1 = normal_buffer[vertexIdx.y];
			optix::float3 n2 = normal_buffer[vertexIdx.z];
			shading_normal = optix::normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
			rtPotentialIntersection(0);
		}
	}
}
///==============
/// BBox Program
///==============
RT_PROGRAM void triangle_mesh_bounds(int primIdx, float result[6])
{
	const uint3 vertexIdx = tindex_buffer[primIdx];

	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];
	const float area = optix::length(optix::cross(v1 - v0, v2 - v0));
	
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
	{
		aabb->invalidate();
	}
}