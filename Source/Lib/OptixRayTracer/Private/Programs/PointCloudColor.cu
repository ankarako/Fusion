#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
// #include <PointCloudIntersection.cuh>

rtDeclareVariable(float, radius, , );
rtDeclareVariable(optix::float3, position, attribute point_position, );
rtDeclareVariable(optix::uchar4, color, attribute point_color, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(int, flat_shaded, , );
rtDeclareVariable(float, culling_plane_pos, , );

rtBuffer<optix::float3> vertex_positions;
rtBuffer<optix::uchar4> vertex_colors;

///=================================
///	Point cloud intersection program
///==================================
RT_PROGRAM void pointcloud_intersect(int primIdx)
{
	// const optix::float3 pos = vertex_positions[primIdx];
	// if (fu::rt::pointcloud_intersection(ray, pos, radius))
	// {
	// 	position = pos;
	// 	color = vertex_colors[primIdx];
	// 	rtReportIntersection(0);
	// }
	const optix::float3 pos = vertex_positions[primIdx];
	if (-pos.y < culling_plane_pos)
	{
		const float rad = radius;
		optix::float3 pos_orig_vec = ray.origin - pos;
		float b = optix::dot(pos_orig_vec, ray.direction);
		float c = optix::dot(pos_orig_vec, pos_orig_vec) - rad * rad;
		float disc = b * b - c;
	
		if (disc > 0.0f)
		{
			float sdisc = sqrtf(disc);
			float root1 = (-b - sdisc);
			bool check_second = true;
			if (rtPotentialIntersection(root1))
			{
				position = pos;
				color = vertex_colors[primIdx];
				if (rtReportIntersection(0))
				{
					check_second = false;
				}
				if (check_second)
				{
					float root2 = -b + sdisc;
					if (rtPotentialIntersection(root2))
					{
						position = pos;
						color = vertex_colors[primIdx];
						rtReportIntersection(0);
					}
				}
			}
		}
	}
}
///===================================
/// Point cloud bounding box program
///===================================
RT_PROGRAM void pointcloud_bounds(int primIdx, float result[6])
{
	const optix::float3 pos = vertex_positions[primIdx];
	const float rad = radius;
	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min.x = pos.x - rad;
	aabb->m_min.y = pos.y - rad;
	aabb->m_min.z = pos.z - rad;
	aabb->m_max.x = pos.x + rad;
	aabb->m_max.y = pos.y + rad;
	aabb->m_max.z = pos.z + rad;
}
///==============================
///	point cloud closest hit
///==============================
RT_PROGRAM void pointcloud_closest_hit()
{
	if (flat_shaded)
	{
		prd_radiance.Result = color;
	}
}
///====================
/// point cloud any hit
///====================
RT_PROGRAM void pointcloud_any_hit()
{
	// rtTerminateRay();
}