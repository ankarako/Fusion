#include "hip/hip_runtime.h"
#include <optix_world.h>

rtBuffer<float3> 	vertex_buffer;
rtBuffer<float3> 	normal_buffer;
rtBuffer<float2>	texcoord_buffer;
rtBuffer<uint3>		tindex_buffer;
rtBuffer<int>		material_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

///==========================================
///	Intersection Program for Triangle Meshes
///==========================================
RT_PROGRAM void triangle_mesh_intersect(int primIdx)
{
	const uint3 vertexIdx = tindex_buffer[primIdx];

	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];

	float3 normal;
	float t;
	float beta;
	float gamma;
	if (optix::intersect_triangle(ray, v0, v1, v2, normal, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			/// FIXME: make this right huh?
			rtReportIntersection(0);
		}
	}
}
///==========================================
///	Bounding Box program for Triangle Meshes
///==========================================
RT_PROGRAM void triangle_mesh_bounds(int primIdx, float result[6])
{
	const uint3 vertexIdx = tindex_buffer[primIdx];

	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];
	const float area = optix::length(optix::cross(v1 - v0, v2 - v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
	{
		aabb->invalidate();
	}
}