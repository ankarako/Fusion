#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>

rtBuffer<float3> 	vertex_buffer;
rtBuffer<float3> 	normal_buffer;
rtBuffer<uchar4>	color_buffer;
rtBuffer<float2>	texcoord_buffer;
rtBuffer<uint3>		tindex_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(optix::float2, cur_texcoord, 		attribute VertexTexcoord, );
rtDeclareVariable(optix::uchar4, cur_color,			attribute TriangleColor, );
rtDeclareVariable(optix::float3, cur_geom_normal,	attribute GeometricNormal, );
rtDeclareVariable(optix::float3, cur_shad_normal,	attribute ShadingNormal, );
rtDeclareVariable(optix::float3, back_hit_point,	attribute BackHitPoint, );
rtDeclareVariable(optix::float3, front_hit_point,	attribute FrontHitPoint, );

///==========================================
///	Intersection Program for Triangle Meshes
///==========================================
RT_PROGRAM void triangle_mesh_intersect(int primIdx)
{
	const uint3 vertexIdx = tindex_buffer[primIdx];

	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];

	float3 normal;
	float t;
	float beta;
	float gamma;
	if (optix::intersect_triangle(ray, v0, v1, v2, normal, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			///==========================
			/// Check for normal buffer
			///==========================
			if (normal_buffer.size() == 0)
			{
				cur_shad_normal = cur_geom_normal;
			}
			else
			{
				optix::float3 n0 = normal_buffer[vertexIdx.x];
				optix::float3 n1 = normal_buffer[vertexIdx.y];
				optix::float3 n2 = normal_buffer[vertexIdx.z];
				/// just for debuggin
				optix::float3 n = (n0 + n1 + n2) / 3;
				cur_geom_normal = optix::normalize(n);
				cur_shad_normal = optix::normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
				optix::float3 world_shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, cur_geom_normal));
				//cur_color = fu::rt::make_color(world_shading_normal);
			}
			///===========================
			/// check for texcoord buffer
			///===========================
			if (texcoord_buffer.size() == 0)
			{
				cur_texcoord = make_float2(0.0f, 0.0f);
			}
			else
			{
				float2 t0 = texcoord_buffer[vertexIdx.x];
				float2 t1 = texcoord_buffer[vertexIdx.y];
				float2 t2 = texcoord_buffer[vertexIdx.z];
				cur_texcoord = t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma);
			}
			///=======================
			/// check for color buffer
			///=======================
			if (color_buffer.size() != 0)
			{
				optix::uchar4 c0 = color_buffer[vertexIdx.x];
				optix::uchar4 c1 = color_buffer[vertexIdx.y];
				optix::uchar4 c2 = color_buffer[vertexIdx.z];
				optix::float3 hitpoint = ray.origin + t * ray.direction;
				cur_color = fu::rt::triangle_color(v0, v1, v2, hitpoint, c0, c1, c2);
			}
			/// material is always zero
			rtReportIntersection(0);
		}
	}
}
///==========================================
///	Bounding Box program for Triangle Meshes
///==========================================
RT_PROGRAM void triangle_mesh_bounds(int primIdx, float result[6])
{
	const uint3 vertexIdx = tindex_buffer[primIdx];

	const float3 v0 = vertex_buffer[vertexIdx.x];
	const float3 v1 = vertex_buffer[vertexIdx.y];
	const float3 v2 = vertex_buffer[vertexIdx.z];

	const float area = optix::length(optix::cross(v1 - v0, v2 - v0));
	
	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
	{
		aabb->invalidate();
	}
}
