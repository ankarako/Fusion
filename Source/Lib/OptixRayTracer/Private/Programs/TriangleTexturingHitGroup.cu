#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtDeclareVariable(optix::float2, 				cur_texcoord, 		attribute VertexTexcoord, );
rtDeclareVariable(optix::float3, 				cur_geom_normal,	attribute GeometricNormal, );
rtDeclareVariable(optix::float3, 				cur_shad_normal,	attribute ShadingNormal, );
rtDeclareVariable(optix::uchar4, 				cur_color,			attribute TriangleColor, );
rtDeclareVariable(fu::rt::PerRayData_Texturing,	prd_texturing, 		rtPayload, );
rtDeclareVariable(optix::Ray,					ray,				rtCurrentRay, );

RT_PROGRAM void closest_hit_texturing()
{
	optix::float3 world_shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, cur_shad_normal));
	optix::float3 world_geometric_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, cur_geom_normal));

	optix::float3 ffnormal = optix::faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
	const float weight = optix::dot(world_geometric_normal, -ray.direction);
	// rtPrintf("w: %f\n", weight);
	prd_texturing.Output.Color =  prd_texturing.PixelValue;
	prd_texturing.Output.Texcoord = cur_texcoord;
	prd_texturing.Output.Weight = weight;
	prd_texturing.Output.CamId = prd_texturing.CameraId;
}

RT_PROGRAM void any_hit()
{
	// rtPrintf("any\n");
	// prd_texturing.Output = { prd_texturing.PixelValue, cur_texcoord, prd_texturing.CameraId, 1.0f };
}