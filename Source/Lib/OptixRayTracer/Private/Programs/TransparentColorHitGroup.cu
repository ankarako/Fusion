#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>


rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable(optix::float4, transparent_color, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, hit_dist, rtIntersectionDistance, );
rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );

RT_PROGRAM void transparent_color_closest_hit()
{
	prd_radiance.Result = fu::rt::make_color(transparent_color);
	optix::float3 hitpoint = ray.origin + hit_dist * ray.direction;
	optix::Ray continuationRay = optix::make_Ray(hitpoint, ray.direction, RadianceRayType, scene_epsilon, RT_DEFAULT_MAX);
	fu::rt::PerRayData_Radiance new_prd;
	rtTrace(top_object, continuationRay, new_prd);
	optix::uchar4 color = new_prd.Result;
	optix::float4 fcolor = 
		optix::make_float4((float)color.x, (float)color.y, (float)color.z, (float)color.w);
	fcolor /= 255.f;
	fcolor = fcolor * (1.0f - transparent_color.w);

	prd_radiance.Result = fu::rt::make_color(fcolor);
}

RT_PROGRAM void transparent_color_any_hit()
{
	//prd_radiance.Result = fu::rt::make_color(transparent_color);
}