#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtBuffer<uchar4>						TextureBuffer;
rtBuffer<optix::uchar4> 				OutputBuffer;
rtTextureSampler<uchar4, 2>				TextureSampler;

rtDeclareVariable(optix::float3, eye, , );
rtDeclareVariable(optix::float3, up, , );
rtDeclareVariable(optix::float3, left, , );
rtDeclareVariable(optix::float3, lookat, , );
rtDeclareVariable(int, camId, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(optix::uint2, launch_idx, rtLaunchIndex, );
rtDeclareVariable(optix::uint2, launch_dims, rtLaunchDim, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(fu::rt::PerRayData_Texturing, prd_texturing, rtPayload, );
rtDeclareVariable(optix::Matrix4x4, extrinsics, , );
rtDeclareVariable(optix::Matrix3x3, intrinsics, , );

RT_PROGRAM void TexturingRaygen()
{
	float u = (float)launch_idx.x / (float)launch_dims.x;
	float v = (float)launch_idx.y / (float)launch_dims.y;
	uchar4 color = tex2D(TextureSampler, u, v);
	fu::rt::PerRayData_Texturing prd_texturing;
	prd_texturing.PixelValue = color;
	prd_texturing.CameraId = camId;
	optix::float3 ray_origin = eye;
	optix::float3 ray_direction = optix::normalize(left * u + up * v + lookat);
	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
	rtTrace(top_object, ray, prd_texturing);
	
}