#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtBuffer<optix::uchar4>					TextureBuffer;
rtTextureSampler<float4, 2>				TextureSampler;

rtBuffer<optix::uchar4, 2>				OutputColorBuffer;
rtBuffer<optix::float2, 2>				OutputTexcoordBuffer;
// // TODO: delete the debug buffer
// rtBuffer<optix::uchar4, 2>				DebugBuffer;
rtBuffer<float, 2>						OutputWeightBuffer;
rtBuffer<int, 2>						OutputCamIdBuffer;

rtDeclareVariable(int, camId, , );
rtDeclareVariable(unsigned int, dim_mult, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(optix::uint2, launch_idx, rtLaunchIndex, );
rtDeclareVariable(optix::uint2, launch_dims, rtLaunchDim, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(fu::rt::PerRayData_Texturing, prd_texturing, rtPayload, );
rtDeclareVariable(optix::Matrix4x4, extrinsics, , );
rtDeclareVariable(optix::Matrix4x4, intrinsics, , );

RT_PROGRAM void TexturingRaygen()
{
	const unsigned int tu = launch_idx.x / dim_mult;
	const unsigned int tv = launch_idx.y / dim_mult;
	// rtPrintf("launch dims: %u x %u\n", launch_dims.x, launch_dims.y);
	// rtPrintf("lidx.x x lidx.y | tu x tv: %u x %u | %u x %u\n", launch_idx.x, launch_idx.y, tu, tv);
	float4 color = tex2D(TextureSampler, tu, tv);
	uchar4 pix = fu::rt::make_color(optix::make_float3(color));

	fu::rt::PerRayData_Texturing prd_texturing;
	prd_texturing.PixelValue = pix;
	prd_texturing.Output = { optix::make_uchar4(0, 0, 0, 0), optix::make_float2(0, 0), 0.0f, camId };
	
	const optix::float4 eye = extrinsics * optix::make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	const float fx = intrinsics.getRow(0).x;
	const float fy = intrinsics.getRow(1).y;
	const float cx = intrinsics.getRow(0).z;
	const float cy = intrinsics.getRow(1).z;

	const float inv_mult = 1.0f / (float)dim_mult;
	const float u = (float)launch_idx.x * inv_mult;
	const float v = (float)launch_idx.y * inv_mult;
	const float x = (u - cx) * 1.0f / fx;
	const float y = (v - cy) * 1.0f / fy;

	const optix::float3 ddir = optix::make_float3(x, y, 1.0f);
	// optix::float4 ss = (intrinsics. * optix::make_float4(ddir, 0.0f));
	const optix::float4 dir = extrinsics * optix::make_float4(optix::normalize(ddir), 0.0f);
	const optix::float3 ray_origin = optix::make_float3(eye);
	const optix::float3 ray_direction = optix::normalize(optix::make_float3(dir));
	
	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
	rtTrace(top_object, ray, prd_texturing);

	OutputColorBuffer[launch_idx] 		= prd_texturing.Output.Color;
	OutputTexcoordBuffer[launch_idx] 	= prd_texturing.Output.Texcoord;
	OutputWeightBuffer[launch_idx] 		= prd_texturing.Output.Weight;
	OutputCamIdBuffer[launch_idx] 		= prd_texturing.Output.CamId;
	// uint2 lidx;
	// lidx.x = lidx.x / dim_mult;
	// lidx.y = lidx.y / dim_mult;
	// DebugBuffer[lidx]  = pix;
	
}