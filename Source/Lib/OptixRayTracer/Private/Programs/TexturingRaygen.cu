#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtBuffer<optix::uchar4>					TextureBuffer;
rtTextureSampler<float4, 2>				TextureSampler;
rtBuffer<fu::rt::TexturingOutput, 2>		TexturingOutputBuffer;


rtDeclareVariable(optix::float3, eye, , );
rtDeclareVariable(optix::float3, up, , );
rtDeclareVariable(optix::float3, left, , );
rtDeclareVariable(optix::float3, lookat, , );
rtDeclareVariable(optix::float3, U, , );
rtDeclareVariable(optix::float3, V, , );
rtDeclareVariable(optix::float3, W, , );
rtDeclareVariable(int, camId, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(optix::uint2, launch_idx, rtLaunchIndex, );
rtDeclareVariable(optix::uint2, launch_dims, rtLaunchDim, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(fu::rt::PerRayData_Texturing, prd_texturing, rtPayload, );
rtDeclareVariable(optix::Matrix4x4, extrinsics, , );
rtDeclareVariable(optix::Matrix3x3, intrinsics, , );

RT_PROGRAM void TexturingRaygen()
{
	float u = (float)launch_idx.x / (float)launch_dims.x * 2.0f - 1.0f;
	float v = (float)launch_idx.y / (float)launch_dims.y * 2.0f - 1.0f;

	
	float4 color = tex2D(TextureSampler, v, u);
	uchar4 pix = fu::rt::make_color(optix::make_float3(color));
	
	fu::rt::PerRayData_Texturing prd_texturing;
	prd_texturing.PixelValue = pix;
	prd_texturing.CameraId = camId;
	prd_texturing.Output = { optix::make_uchar4(0, 0, 0, 0), optix::make_float2(0, 0), 0.0f, camId };

	optix::float3 ray_origin = eye;

	float fx = intrinsics.getRow(0).x;
	float fy = intrinsics.getRow(1).y;
	fx = fx / (float)launch_dims.x;
	fy = fy / (float)launch_dims.y;
	float4 u_dir = extrinsics * make_float4(u, 0.0f, 0.0f, 1.0f);
	float4 v_dir = extrinsics * make_float4(0.0f, v, 0.0f, 1.0f);
	float3 uu = make_float3(u_dir.x / u_dir.w, u_dir.y / u_dir.w, u_dir.z / u_dir.w);
	float3 vv = make_float3(v_dir.x / v_dir.w, v_dir.y / v_dir.w, v_dir.z / v_dir.w);
	
	optix::float3 ray_direction = optix::normalize(u * U + v * V + W);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
	rtTrace(top_object, ray, prd_texturing);

	TexturingOutputBuffer[launch_idx] = prd_texturing.Output;
}