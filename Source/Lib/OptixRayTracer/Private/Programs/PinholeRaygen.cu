#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>


rtDeclareVariable(optix::float3, eye, , );
rtDeclareVariable(optix::float3, U, , );
rtDeclareVariable(optix::float3, V, , );
rtDeclareVariable(optix::float3, W, , );

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, ,);
rtDeclareVariable(optix::uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(optix::uint2, launch_dims, rtLaunchDim, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );

rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void PinholeRaygen()
{
	optix::float2 d = make_float2(launch_index) / make_float2(launch_dims) * 2.0f - 1.0f;
	optix::float3 ray_origin = eye;
	optix::float3 ray_direction = optix::normalize(d.x * U + d.y * V + W);
	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RadianceRayType, scene_epsilon, RT_DEFAULT_MAX);
	fu::rt::PerRayData_Radiance prd;
	prd.Importance = 1.0f;
	prd.Depth = 0;
	rtTrace(top_object, ray, prd);
	output_buffer[launch_index] = prd.Result;
}