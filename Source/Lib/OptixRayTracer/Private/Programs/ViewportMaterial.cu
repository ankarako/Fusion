#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::float3, texcoord, attribute texcoord, );

rtTextureSampler<float4, 2> ViewportTextureSampler;
RT_PROGRAM void viewport_closest_hit()
{
	const float u = texcoord.x;
	const float v = texcoord.y;
	const float4 color = tex2D(ViewportTextureSampler, u, v);
	prd_radiance.Result = fu::rt::make_color(optix::make_float3(color));
}
