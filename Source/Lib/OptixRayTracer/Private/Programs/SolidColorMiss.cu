#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <RayPayload.cuh>
#include <Helpers.cuh>

rtDeclareVariable(fu::rt::PerRayData_Radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, solid_color, , );
///==============================================
/// Simple miss program
/// puts a specified solid color when ray misses
///==============================================
RT_PROGRAM void SolidColorMiss()
{
	prd_radiance.Result = fu::rt::make_color(solid_color);
}