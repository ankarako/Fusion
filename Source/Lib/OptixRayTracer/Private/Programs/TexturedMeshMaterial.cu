#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtBuffer<uchar4>			TextureBuffer;
rtTextureSampler<float4, 2> TextureSampler;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(optix::float2, cur_texcoord, 		attribute VertexTexcoord, );
rtDeclareVariable(optix::uchar4, cur_color,			attribute TriangleColor, );
rtDeclareVariable(optix::float3, cur_geom_normal,	attribute GeometricNormal, );
rtDeclareVariable(optix::float3, cur_shad_normal,	attribute ShadingNormal, );
rtDeclareVariable(optix::float3, back_hit_point,	attribute BackHitPoint, );
rtDeclareVariable(optix::float3, front_hit_point,	attribute FrontHitPoint, );
rtDeclareVariable(fu::rt::PerRayData_Radiance,	prd_radiance, 		rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
	float u = cur_texcoord.x;
	float v = cur_texcoord.y;
	float4 color = tex2D(TextureSampler, u, v);
	prd_radiance.Result = fu::rt::make_color(optix::make_float3(color));
}

RT_PROGRAM void any_hit()
{

}