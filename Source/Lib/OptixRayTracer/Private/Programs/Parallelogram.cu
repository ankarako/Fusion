#include "hip/hip_runtime.h"
#include <optix_world.h>
/// inputs
rtDeclareVariable(optix::float4, plane, , );
rtDeclareVariable(optix::float3, v1, , );
rtDeclareVariable(optix::float3, v2, , );
rtDeclareVariable(optix::float3, anchor, , );
rtDeclareVariable(int, lgt_instance, , ) = { 0 };
/// attributes
rtDeclareVariable(optix::float3, texcoord, attribute texcoord, );
rtDeclareVariable(optix::float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(optix::float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void parallelogram_intersect(int primIdx)
{
	optix::float3 n = optix::make_float3(plane);
	float dt = optix::dot(ray.direction, n);
	float t = (plane.w - optix::dot(n, ray.origin)) / dt;
	
	if (t > ray.tmin && t < ray.tmax)
	{
		optix::float3 p = ray.origin + t * ray.direction;
		optix::float3 vi = p - anchor;
		float a1 = optix::dot(v1, vi);
		
		if (a1 >= 0 && a1 <= 1)
		{
			float a2 = optix::dot(v1, vi);

			if (a2 >= 0 && a2 <= 1)
			{
				if (rtPotentialIntersection(t))
				{
					shading_normal = geometric_normal = n;
					texcoord = optix::make_float3(a1, a2, 0);
					lgt_idx = lgt_instance;
					rtReportIntersection(0);
				}
			}
		}
	}
}

RT_PROGRAM void parallelogram_bounds(int, float result[6])
{
	const optix::float3 tv1 = v1 / optix::dot(v1, v1);
	const optix::float3 tv2 = v2 / optix::dot(v2, v2);
	const optix::float3 p00 = anchor;
	const optix::float3 p01 = anchor + tv1;
	const optix::float3 p10 = anchor + tv2;
	const optix::float3 p11 = anchor + tv1 + tv2;
	const float area = optix::length(optix::cross(tv1, tv2));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(p00, p01), fminf(p10, p11));
		aabb->m_max = fmaxf(fmaxf(p00, p01), fmaxf(p10, p11));
	}
	else
	{
		aabb->invalidate();
	}
}