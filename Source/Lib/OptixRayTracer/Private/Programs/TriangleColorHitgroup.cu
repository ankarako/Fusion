#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>
#include <RayPayload.cuh>

rtDeclareVariable(optix::float3, 				cur_geom_normal,	attribute GeometricNormal, );
rtDeclareVariable(optix::float3, 				cur_shad_normal,	attribute ShadingNormal, );
rtDeclareVariable(optix::uchar4, 				cur_color,			attribute TriangleColor, );
rtDeclareVariable(fu::rt::PerRayData_Radiance,	prd_radiance, 		rtPayload, );
rtDeclareVariable(optix::Ray,					ray,				rtCurrenRay, );

RT_PROGRAM void closest_hit_radiance()
{
	optix::float3 world_shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, cur_shad_normal));
	optix::float3 world_geometric_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, cur_shad_normal));

	optix::float3 ffnormal = optix::faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
	
	prd_radiance.Result = cur_color;
}

RT_PROGRAM void any_hit()
{
	rtTerminateRay();
}