#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <Helpers.cuh>

rtBuffer<optix::float3> vertex_buffer;
rtDeclareVariable(optix::float3, color, attribute quad_color, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void quad_intersect(int primIdx)
{
	
	optix::float3 v0 = vertex_buffer[0];
	optix::float3 v1 = vertex_buffer[1];
	optix::float3 v2 = vertex_buffer[2];
	optix::float3 v3 = vertex_buffer[3];

	optix::float3 normal;
	float t;
	float beta;
	float gamma;
	if (optix::intersect_triangle(ray, v0, v1, v3, normal, t, beta, gamma) || optix::intersect_triangle(ray, v0, v2, v3, normal, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			rtReportIntersection(0);
		}
		
	}
}

RT_PROGRAM void quad_bounds(int primIdx, float result[6])
{
	const optix::float3 v0 = vertex_buffer[0];
	const optix::float3 v1 = vertex_buffer[1];
	const optix::float3 v2 = vertex_buffer[2];
	const optix::float3 v3 = vertex_buffer[3];

	optix::Aabb* aabb = (optix::Aabb*)result;

	aabb->m_min = fminf(fminf(v0, v1), fminf(v2, v3));
	aabb->m_max = fmaxf(fmaxf(v0, v1), fmaxf(v2, v3));
}